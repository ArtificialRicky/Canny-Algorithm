#include "hip/hip_runtime.h"
// gradient_image.cu
#include "gradient_image.h"
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/core.hpp>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>


#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

// 定义角度阈值判断函数
__device__ bool is_45(float Angle) {
    return (Angle > 0 && Angle <= 45) || (Angle > 180 && Angle <= 225);
}

__device__ bool is_90(float Angle) {
    return (Angle > 45 && Angle <= 90) || (Angle > 225 && Angle <= 270);
}

__device__ bool is_135(float Angle) {
    return (Angle > 90 && Angle <= 135) || (Angle > 270 && Angle <= 315);
}

__device__ bool is_180(float Angle) {
    return (Angle == 0) || (Angle > 135 && Angle <= 180) || (Angle > 315 && Angle <= 360);
}

void Gradient_image(const cv::Mat &img_src,
    cv::Mat &img_out,      // an empty matrix to store result
    cv::Mat_<float> &angle)     // an empty matrix to store arctan(Gy / Gx)
{
    angle = cv::Mat_<float>::zeros(img_src.size());
    img_out = cv::Mat::zeros(img_src.size(), CV_8UC1);
    int row_minus_1 = img_src.rows - 1;
    int col_minus_1 = img_src.cols - 1;

    // int row = img_src.rows;
    // int col = img_src.cols;

    auto point = img_src.data;
    int step = img_src.step;

    for (int i = 1; i < row_minus_1; ++i) {
        for (int j = 1; j < col_minus_1; ++j) {
            uchar pixel_00 = point[(i - 1) * step + j - 1];
            uchar pixel_01 = point[(i - 1) * step + j];
            uchar pixel_02 = point[(i - 1) * step + j + 1];
            uchar pixel_10 = point[i * step + j - 1];
            // uchar pixel_11 = point[i * step + j];
            uchar pixel_12 = point[i * step + j + 1];
            uchar pixel_20 = point[(i + 1) * step + j - 1];
            uchar pixel_21 = point[(i + 1) * step + j];
            uchar pixel_22 = point[(i + 1) * step + j + 1];

            // float grad_x = (-1 * pixel_00) + (-2 * pixel_10) + (-1 * pixel_20) + (1 * pixel_02) + (2 * pixel_12) + (1 * pixel_22);
            float grad_x = pixel_02 + (2 * pixel_12) + pixel_22 - pixel_00 - (2 * pixel_10) - pixel_20;

            // float grad_y = (1 * pixel_00) + (2 * pixel_01) + (1 * pixel_02) + (-1 * pixel_20) + (-2 * pixel_21) + (-1 * pixel_22);
            float grad_y = pixel_00 + (2 * pixel_01) + pixel_02 - pixel_20 - (2 * pixel_21) - pixel_22;

            angle.at<float>(i, j) = atan(grad_y / (grad_x == 0 ? 0.00001 : grad_x));
            img_out.at<uchar>(i, j) = sqrt(grad_x * grad_x + grad_y * grad_y);
        }
    }
}

// CUDA 核函数：每个线程处理图像中一个非边界像素
__global__ void GradientImageKernel(const unsigned char* src, 
                                      unsigned char* dst, 
                                      float* angle, 
                                      int rows, int cols, 
                                      int step)
{
    // 计算线程对应的 (i, j) 像素位置
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    // 忽略边界像素，避免越界
    if (i < 1 || i >= rows - 1 || j < 1 || j >= cols - 1)
        return;
    
    int index = i * step + j;

    // 读取 3×3 邻域内的像素
    unsigned char pixel_00 = src[(i - 1) * step + (j - 1)];
    unsigned char pixel_01 = src[(i - 1) * step + j];
    unsigned char pixel_02 = src[(i - 1) * step + (j + 1)];
    unsigned char pixel_10 = src[i * step + (j - 1)];
    unsigned char pixel_12 = src[i * step + (j + 1)];
    unsigned char pixel_20 = src[(i + 1) * step + (j - 1)];
    unsigned char pixel_21 = src[(i + 1) * step + j];
    unsigned char pixel_22 = src[(i + 1) * step + (j + 1)];

    // 使用 Sobel 算子计算水平和垂直梯度
    float grad_x = pixel_02 + (2 * pixel_12) + pixel_22 - pixel_00 - (2 * pixel_10) - pixel_20;
    float grad_y = pixel_00 + (2 * pixel_01) + pixel_02 - pixel_20 - (2 * pixel_21) - pixel_22;
    
    // 计算梯度幅值
    float grad = sqrtf(grad_x * grad_x + grad_y * grad_y);
    dst[index] = (unsigned char)grad;
    
    // 计算梯度方向，使用 atan2f，更稳健；当 grad_x == 0 时，用极小值代替
    angle[i * cols + j] = atan2f(grad_y, (grad_x == 0 ? 0.00001f : grad_x));
}

// 主机端 CUDA 包装函数：调用核函数计算梯度
void Gradient_image_cuda(const cv::Mat &img_src,
                         cv::Mat &img_out,
                         cv::Mat_<float> &angle)
{
    int rows = img_src.rows;
    int cols = img_src.cols;
    size_t step = img_src.step; // 每行字节数

    // 初始化输出图像和角度矩阵
    img_out.create(rows, cols, CV_8UC1);
    angle.create(rows, cols);

    // 申请设备内存
    unsigned char *d_src = nullptr, *d_dst = nullptr;
    float *d_angle = nullptr;
    size_t img_size = rows * step;

    hipMalloc((void**)&d_src, img_size);
    hipMalloc((void**)&d_dst, img_size);
    hipMalloc((void**)&d_angle, rows * cols * sizeof(float));

    // 将输入图像数据复制到设备内存
    hipMemcpy(d_src, img_src.data, img_size, hipMemcpyHostToDevice);

    // 定义线程块和网格尺寸
    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

    // 启动 CUDA 核函数
    GradientImageKernel<<<grid, block>>>(d_src, d_dst, d_angle, rows, cols, step);
    hipDeviceSynchronize();

    // 将结果复制回主机内存
    hipMemcpy(img_out.data, d_dst, img_size, hipMemcpyDeviceToHost);
    hipMemcpy(angle.data, d_angle, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_angle);
}

// 非极大值抑制：仅保留局部极大值
__global__ void non_maximum_suppression_kernel(
    unsigned char* img, const float* angle, int rows, int cols, int step)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < 1 || j < 1 || i >= rows - 1 || j >= cols - 1) return;

    int idx = i * step + j;
    float Angle = angle[i * cols + j];
    uchar value = img[idx];
    uchar previous = 0, next = 0;

    if (is_45(Angle)) {
        previous = img[(i - 1) * step + (j + 1)];
        next = img[(i + 1) * step + (j - 1)];
    } else if (is_90(Angle)) {
        previous = img[(i - 1) * step + j];
        next = img[(i + 1) * step + j];
    } else if (is_135(Angle)) {
        previous = img[(i - 1) * step + (j - 1)];
        next = img[(i + 1) * step + (j + 1)];
    } else if (is_180(Angle)) {
        previous = img[i * step + (j - 1)];
        next = img[i * step + (j + 1)];
    }

    if (value < previous || value < next)
        img[idx] = 0;
}

void non_maximum_suppression(cv::Mat &img_out, const cv::Mat_<float> &angle)
{
    int rows = img_out.rows;
    int cols = img_out.cols;
    int step = img_out.step;

    size_t img_size = rows * step * sizeof(uchar);
    size_t angle_size = rows * cols * sizeof(float);

    uchar *d_img;
    float *d_angle;

    hipMalloc(&d_img, img_size);
    hipMalloc(&d_angle, angle_size);

    hipMemcpy(d_img, img_out.data, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_angle, angle.ptr<float>(), angle_size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
    non_maximum_suppression_kernel<<<grid, block>>>(d_img, d_angle, rows, cols, step);
    hipDeviceSynchronize();

    hipMemcpy(img_out.data, d_img, img_size, hipMemcpyDeviceToHost);
    hipFree(d_img);
    hipFree(d_angle);
}

__global__ void double_threshold_kernel(uchar *img, int width, int height, int low, int high, int step)
{
    // 注意：这个版本假设核函数只处理内部区域，不访问图像边界像素
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    // 为了使用简单边界，分配 tile 比实际 block 大两列两行
    __shared__ uchar tile[BLOCK_SIZE_Y + 2][BLOCK_SIZE_X + 2];

    int local_x = threadIdx.x + 1;
    int local_y = threadIdx.y + 1;

    // 加载中心区域
    tile[local_y][local_x] = img[y * step + x];

    // 保证共享内存加载完毕
    __syncthreads();

    // 仅处理内部区域像素，不处理边界（已通过传参或 grid 配置排除）
    uchar value = tile[local_y][local_x];
    if (value < low) {
        value = 0;
    } else if (value > high) {
        value = 255;
    } else {
        bool has_strong_neighbor = false;
        #pragma unroll
        for (int m = -1; m <= 1 && !has_strong_neighbor; ++m) {
            #pragma unroll
            for (int n = -1; n <= 1; ++n) {
                if (m == 0 && n == 0) continue;
                if (tile[local_y + m][local_x + n] > high) {
                    value = 255;
                    has_strong_neighbor = true;
                    break;
                }
            }
        }
        if (!has_strong_neighbor)
            value = 0;
    }

    // 写回结果到全局内存
    img[y * step + x] = value;
}

// 双阈值处理：根据低/高阈值确定边缘像素
void double_threshold(cv::Mat &img_out, const int &low, const int &high) {
    assert(low >= 0 && high >= 0 && low <= high);
    assert(img_out.type() == CV_8UC1);

    uchar *d_img;
    size_t img_size = img_out.rows * img_out.step;

    hipMalloc(&d_img, img_size);
    hipMemcpy(d_img, img_out.data, img_size, hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid((img_out.cols + block.x - 1) / block.x,
              (img_out.rows + block.y - 1) / block.y);

    double_threshold_kernel<<<grid, block>>>(d_img, img_out.cols, img_out.rows, low, high, img_out.step);
    hipDeviceSynchronize();

    hipMemcpy(img_out.data, d_img, img_size, hipMemcpyDeviceToHost);
    hipFree(d_img);
}

// Canny 边缘检测：调用 CUDA 计算梯度，然后进行非极大值抑制和双阈值处理
void Canny(const cv::Mat &img_src, cv::Mat &img_out,
           const int &low_threshold, const int &high_threshold) 
{
    assert(low_threshold <= high_threshold);
    cv::Mat_<float> angle;

    int64 t1 = cv::getTickCount();
    Gradient_image_cuda(img_src, img_out, angle);
    // Gradient_image(img_src, img_out, angle);
    int64 t2 = cv::getTickCount();
    std::cout << "Gradient_image_cuda: " << (t2 - t1) / cv::getTickFrequency() << " sec\n";

    non_maximum_suppression(img_out, angle);
    int64 t3 = cv::getTickCount();
    std::cout << "non_maximum_suppression: " << (t3 - t2) / cv::getTickFrequency() << " sec\n";

    double_threshold(img_out, low_threshold, high_threshold);
    int64 t4 = cv::getTickCount();
    std::cout << "double_threshold: " << (t4 - t3) / cv::getTickFrequency() << " sec\n";

    std::cout << "Total Canny time: " << (t4 - t1) / cv::getTickFrequency() << " sec\n";
}